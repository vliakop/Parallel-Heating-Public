#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include ""



#define NXPROB      20                 /* x dimension of problem grid */
#define NYPROB      20                 /* y dimension of problem grid */
#define STEPS       100                /* number of time steps */

struct Parms { 
  float cx;
  float cy;
} parms = {0.1, 0.1};


/**************************************************************************
 *  subroutine initdat
 ****************************************************************************/
__global__ void inidat(int *d_dimensions, float *d_u){

	int pos = (blockIdx.x * blockDim.x) + threadIdx.x;
	d_u[pos] = (float)(blockIdx.x*((*d_dimensions) - blockIdx.x - 1)*threadIdx.x*((*d_dimensions) - threadIdx.x - 1));
	__syncthreads();

}

/**************************************************************************
 *  subroutine prdat
 ****************************************************************************/
	
	void prdat(int X, int Y, float u[NXPROB][NYPROB], char *filename){
		
		FILE *fp;
		fp = fopen(filename, "w+");
		if(fp == NULL){
			printf("Couldn't open %s\n", filename);
			return;
		}
		int x, y;
		for (x = 0; x < X; x++){
			for(y = 0; y < Y; y++){
				fprintf(fp, "%6.1lf", u[x][y]);
				if(y == X - 1){
					fprintf(fp, "\n");
				}
				else{
					fprintf(fp, " ");
				}
			}
		}
		return;
	}

/**************************************************************************
 *  subroutine update
 ****************************************************************************/
__global__ void update(int *d_dimensions, float *d_cx, float *d_cy, float *d_u1, float *d_u2) {
	
	int index;
	index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(blockIdx.x == 0 || threadIdx.x == 0 || blockIdx.x == (*d_dimensions - 1) || threadIdx.x == (*d_dimensions - 1)){ /* ! on last thread-clause */
		d_u2[index] = 0.0;
		return;
	}
	d_u2[index] = d_u1[index] + (*d_cx) * (d_u1[threadIdx.x + (blockIdx.x + 1)* blockDim.x] + 
					d_u1[threadIdx.x + (blockIdx.x - 1)* blockDim.x] - 2.0 * d_u1[index]) + 
					(*d_cy) * (d_u1[(threadIdx.x + 1) + blockIdx.x * blockDim.x] +
					d_u1[(threadIdx.x - 1) + blockIdx.x * blockDim.x] - 2.0 * d_u1[index]);
	__syncthreads();
	return;
}



int main (int argc, char *argv[]){

	/* host declerations */
	int	iz, it, dimensions, arraySize;
	float  u[2][NXPROB][NYPROB];     /* array for grid */
	
	dimensions = NXPROB;
	arraySize = NXPROB*NYPROB*sizeof(float);

	
	/* device declerations */
	int *d_dimensions;
	float *d_cx, *d_cy;
	float *d_u[2];
	
	/* Device Memory Allocation */
	hipMalloc((void **)&d_dimensions, sizeof(int)); /* X, Y dimensions */
	hipMalloc((void **)&d_cx, sizeof(float));		 /* Parameter 1 */
	hipMalloc((void **)&d_cy, sizeof(float));		 /* Parameter 2*/
	hipMalloc((void **)&d_u[0], arraySize); 		 /* array at time t */
	hipMalloc((void **)&d_u[1], arraySize);		 /* array at time t+1 */

	/* Device memcpy */
	hipMemcpy(d_dimensions, &dimensions, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_cx, &parms.cx, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_cy, &parms.cy, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_u[0], u[0], arraySize, hipMemcpyHostToDevice);
	
	/* Initialise the table */
	printf("Grid size: X= %d  Y= %d  Time steps= %d\n",NXPROB,NYPROB,STEPS);
	printf("Initializing grid and writing initial.dat file...\n");
	inidat << <NXPROB, NYPROB >> >(d_dimensions, d_u[0]);
	hipMemcpy(u[0], d_u[0], arraySize, hipMemcpyDeviceToHost);
	prdat(NXPROB, NYPROB, u[0], "init.dat");
	
	/* Calculate d_u[1] */
	iz = 0;
	for (it = 0; it < STEPS; it++){
		update << <NXPROB, NYPROB >> >(d_dimensions, d_cx, d_cy, d_u[iz], d_u[1 - iz]);
		iz = 1 - iz;
	}
	
	hipMemcpy(u[1], d_u[1], arraySize, hipMemcpyDeviceToHost);
	
	/* Write final output, call X graph and finalize MPI */
	printf("Writing final.dat file, which was calculated using CUDA C, and generating graph...\n");
	prdat(NXPROB, NYPROB, u[1], "final.dat");
	
	hipFree(d_dimensions);
	hipFree(d_cx);
	hipFree(d_cy);
	hipFree(d_u[0]);
	hipFree(d_u[1]);
	return 0;
 }  
